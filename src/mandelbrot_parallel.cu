#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include "lodepng.h"
#include "mandelbrot_parallel.h"

typedef unsigned int uint;
typedef unsigned char uchar;

// Linear interpolation
#define LERP(x, y, t) \
    ((1-t)*x + t*y)

const uint BYTES_PER_PIXEL = 3;
const uint LIMIT           = (1 << 16);

__global__ void render(uchar* image,
                       uint width,
                       uint height,
                       uint max_iterations,
                       float x_scale,
                       float y_scale,
                       float x_adjust,
                       float y_adjust)
{
    uint x_dim = blockIdx.x*blockDim.x + threadIdx.x;
    uint y_dim = blockIdx.y*blockDim.y + threadIdx.y;
    uint index = BYTES_PER_PIXEL * (width * y_dim + x_dim);
    float x_origin = ((float) x_dim/width)*x_scale - x_adjust;
    float y_origin = ((float) y_dim/width)*y_scale - y_adjust;

    float x = 0.0;
    float y = 0.0;
    uint iteration = 0;

    // Escape time algorithm
    while(x*x + y*y < LIMIT && iteration < max_iterations) {
        float xtemp = x*x - y*y + x_origin;
        y = 2*x*y + y_origin;
        x = xtemp;
        iteration++;
    }

    if (iteration == max_iterations) {
        image[index]     = 0;
        image[index + 1] = 0;
        image[index + 2] = 0;
    }
    else {
        // Continous coloring
        // Computes the color as a linear interpolation of surrounding points
        // smoothing the color transition.
        float zn  = sqrt(x*x + y*y);
        float nu  = log10f(log10f(zn) / log10f(2)) / log10f(2);
        float itr = ((float) iteration) + 1 - nu;
        float t   = fmodf(itr, 1.0);
        iteration = (uint) itr;
        uint color1 = iteration;
        uint color2 = iteration+1;

        // Assign RGB values by multiplying the iteration count by a even multiples.
        // Should replace this with a predefined colormap.
        image[index]     = ((uint) LERP(color1, color2, t)*2) % max_iterations;
        image[index + 1] = ((uint) LERP(color1, color2, t)*4) % max_iterations;
        image[index + 2] = ((uint) LERP(color1, color2, t)*6) % max_iterations;
    }
}

void run_mandelbrot_parallel(uchar** out_image,
                             size_t* image_size,
                             uint width, uint height,
                             uint max_iterations,
                             uint block_dim_x, uint block_dim_y,
                             float x_scale,
                             float y_scale,
                             float x_adjust,
                             float y_adjust)
{
    size_t buffer_size = sizeof(uchar) * width * height * BYTES_PER_PIXEL;
    uchar* image;
    hipMalloc((void**) &image, buffer_size);
    uchar* host_image = (uchar*) malloc(buffer_size);

    // Render image
    dim3 blockDim(block_dim_x, block_dim_y);
    dim3 gridDim(width / blockDim.x, height / blockDim.y);
    render<<<gridDim, blockDim>>>(image, width, height, max_iterations,
                                  x_scale, y_scale,
                                  x_adjust, y_adjust);

    // Copy device image to host and output to png file
    hipMemcpy(host_image, image, buffer_size, hipMemcpyDeviceToHost);
    hipFree(image);

    // Output image and size
    *out_image = host_image;
    *image_size = buffer_size;
}

#ifdef STANDALONE

// Scales the x-axis and y-axis, centers the mandelbrot
const float X_SCALE = 3.25;
const float Y_SCALE = 2.5;
const float X_ADJUST = 2;
const float Y_ADJUST = 1.25;

void save_mandelbrot(const char* file_name,
                     uint width, uint height,
                     uint max_iterations,
                     uint block_dim_x, uint block_dim_y)
{
    uchar *host_image;
    size_t image_size;
    run_mandelbrot_parallel(&host_image, &image_size,
                            width, height, max_iterations,
                            block_dim_x, block_dim_y,
                            X_SCALE, Y_SCALE,
                            X_ADJUST, Y_ADJUST);
    lodepng_encode24_file(file_name, host_image, width, height);
    free(host_image);
}

int main(int argc, const char* argv[]) {

    if (argc != 7) {
        fprintf(stderr, "%s [width] [height] [max_iterations] [block_dim_x] [block_dim_y] [file name]\n", argv[0]);
        return 1;
    }
    
    uint width = (uint) atoi(argv[1]);
    uint height = (uint) atoi(argv[2]);
    uint max_iterations = (uint) atoi(argv[3]);
    uint block_dim_x = (uint) atoi(argv[4]);
    uint block_dim_y = (uint) atoi(argv[5]);
    const char* file_name = argv[6];
    save_mandelbrot(file_name,
                    width, height,
                    max_iterations,
                    block_dim_x, block_dim_y);
    return 0;
}

#endif